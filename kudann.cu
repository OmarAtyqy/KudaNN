#include "hip/hip_runtime.h"
#include "kudann.cuh"


// structure to help in the sorting of the distances
typedef struct dist_index
{
    float class_;
    float distance;

} distIndex;

// structure to hold the number of votes for each class
typedef struct num_votes
{
    float class_;
    int votes;

} numVotes;

// helper function to compute the euclidian distance between two vector arrays
__device__ float euclidianDistance(float *a, float *b, int width) {
    float res = 0;
    for (int i = 0; i < width; i++) {
        res = res + (a[i] - b[i])*(a[i] - b[i]);
    }
    return sqrtf(res);
}


__global__ void kudaNN(float *data, float *labels, int data_n_examples, float *predict, int predict_n_examples, int n_features, int n_classes, int k, float *result)
{

    // array to hold all (test, train) pair distances
    extern __shared__ distIndex distances[];

    // thread mapping
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    // use the current thread to compute the euclidian distance between test[idx] and train[idy]
    if (idx < predict_n_examples && idy < data_n_examples)
    {

        // put the values of the current training sample and prediction sample into arrays to pass to the euclidianDistance function
        float *test_row = (float *)malloc(n_features * sizeof(float));
        float *data_row = (float *)malloc(n_features * sizeof(float));
        for (int i = 0; i < n_features; i++)
        {
            test_row[i] = predict[idx * n_features + i];
            data_row[i] = data[idy * n_features + i];
        }

        // compute the euclidian distance
        float distance = euclidianDistance(data_row, test_row, n_features);

        // create a distIndex struct to hold the distance and the class of corresponding example
        distIndex temp;
        temp.distance = distance;
        temp.class_ = labels[idy];

        // assign the distance in the array using the formula (idx, idy) -> idx * n_data + idy
        distances[idx * data_n_examples + idy] = temp;

        // freeing memory
        free(data_row);
        free(test_row);
    }

    // wait for all threads to finish execution
    __syncthreads();

    // after all comparaisons have been made, find the predictions for each result
    if (idx < predict_n_examples)
    {
        // sort the distances that correspond to test example idx increasingly
        // this program uses bubblesort
        // TODO: use a better sorting algorithm for further optimization
        for (int i = 0; i < data_n_examples - 1; i++)
        {
            for (int j = 0; j < data_n_examples - i - 1; j++)
            {
                if (distances[idx * data_n_examples + j].distance > distances[idx * data_n_examples + j + 1].distance)
                {
                    distIndex temp = distances[idx * data_n_examples + j];
                    distances[idx * data_n_examples + j] = distances[idx * data_n_examples + j + 1];
                    distances[idx * data_n_examples + j + 1] = temp;
                }
            }
        }

        // make a prediction using a majority system on the k nearest neighbours
        numVotes *votes = (numVotes *)malloc(sizeof(numVotes) * n_classes);
        for (int i = 0; i < n_classes; i++)
        {
            numVotes vote;
            vote.class_ = i;
            vote.votes = 0;
            votes[i] = vote;
        }

        // run through the first k elements and register their vote
        for (int i = 0; i < k; i++)
        {
            for (int j = 0; j < n_classes; j++)
            {
                if (votes[j].class_ == distances[idx * data_n_examples + i].class_)
                {
                    votes[j].votes++;
                }
            }
        }

        // find the class with the most votes
        float max = 0;
        for (int i = 0; i < k; i++)
        {
            if (votes[i].votes > max)
            {
                // choose the class with the most votes as the prediction
                max = votes[i].votes;
                result[idx] = votes[i].class_;
            }
        }

        // free memory
        free(votes);
    }

    // wait for all threads to finish execution
    __syncthreads();

    // free memory
    free(distances);
}